#include "hip/hip_runtime.h"
/******************************************************************************
MIT License

Copyright (c) 2016 Antti-Pekka Hynninen
Copyright (c) 2016 Oak Ridge National Laboratory (UT-Batelle)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*******************************************************************************/

//
// Micro-benchmark for memory operations
//

#include <cstdio>
#include "CudaUtils.h"

__global__ void clearCacheKernel(int* buffer, const int bufferSize) {
  for (int t=threadIdx.x + blockIdx.x*blockDim.x;t < bufferSize;t+=blockDim.x*gridDim.x) {
    buffer[t] = t;
  }
}

template<typename T>
__global__ void memoryTransactionKernel(T* buffer) {
  __shared__ T sh_a;
  int i = threadIdx.x;
  long long int start = clock64();  
  T a = buffer[i];
  sh_a = a;
  long long int end = clock64();
  printf("%d %f\n", (int)(end - start), (float)sh_a);
}

template <typename T, int niter>
__global__ void pChaseKernel(T* array) {
  
  __shared__ int duration[niter];
  __shared__ T dummy[niter];

  {
    T j = threadIdx.x*32;
    for (int it=0;it < niter;it++) {
      int start = clock();
      j = array[j];
      dummy[it] = j;
      int end = clock();
      duration[it] = end - start;
    }
  }

  if (threadIdx.x == 0) {
    int total_duration = 0;
    int total_duration2 = 0;
    int total_dummy = 0;
    for (int it=1;it < niter;it++) {
      int d = duration[it];
      total_duration += d;
      total_duration2 += d*d;
      total_dummy += (int)dummy[it];
    }
    float avg_duration = (float)total_duration/(float)(niter - 1);
    float avg_duration2 = (float)total_duration2/(float)(niter - 1);
    float std_duration = sqrtf(avg_duration2 - avg_duration*avg_duration);
    printf("%1.2f %1.2f %d\n", avg_duration, std_duration, total_dummy);
    // for (int it=0;it < niter;it++) {
    //   printf("%d %d\n", duration[it], (int)dummy[it]);
    // }
  }
}

template <typename T>
__global__ void pChaseMaxwellKernel(T* array, const int niter) {
  
  extern __shared__ T dummy[];

  int start = clock();
  T j = threadIdx.x*32;
  for (int it=0;it < niter;it++) {
    j = array[j];
    dummy[it] = j;
  }
  int end = clock();
  int duration = (int)(end - start);

  if (threadIdx.x == 0) {
    int total_dummy = 0;
    for (int it=0;it < niter;it++) total_dummy += dummy[it];
    printf("%1.2f %d\n", (float)duration/(float)niter, total_dummy);
  }
}

template <typename T>
__global__ void memoryLatencyKernel(T* bufferIn, T* bufferOut) {
  extern __shared__ int shCycles[];
  // int p = threadIdx.x*(128/sizeof(T)) + 1 + blockIdx.x*1024;
  int p = threadIdx.x;
  //if (threadIdx.x % 32 > 0) p += 128;
  long long int start = clock64();
  T a = bufferIn[p];
  long long int end = clock64();
  shCycles[threadIdx.x] = (int)(end - start);
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    int minCycle = (1 << 30);
    int maxCycle = 0;
    int aveCycle = 0;
    for (int i=0;i < blockDim.x;i++) {
      minCycle = min(minCycle, shCycles[i]);
      maxCycle = max(maxCycle, shCycles[i]);
      aveCycle += shCycles[i];
    }
    printf("%d %d %d\n", minCycle, maxCycle, aveCycle/blockDim.x);
  }
  bufferOut[threadIdx.x] = a;
}

template <typename T>
__global__ void memoryLatencyKernel2(T* bufferOut) {
  extern __shared__ int shCycles[];
  int p = threadIdx.x + 1;
  long long int start = clock64();
  bufferOut[p] = 1.2;
  long long int end = clock64();
  shCycles[threadIdx.x] = (int)(end - start);
  __syncthreads();
  if (blockIdx.x == 0 && threadIdx.x == 0) {
    int minCycle = (1 << 30);
    int maxCycle = 0;
    int aveCycle = 0;
    for (int i=0;i < blockDim.x;i++) {
      minCycle = min(minCycle, shCycles[i]);
      maxCycle = max(maxCycle, shCycles[i]);
      aveCycle += shCycles[i];
    }
    printf("%d %d %d\n", minCycle, maxCycle, aveCycle/blockDim.x);
  }
}

// __global__ void memoryReadKernel(int* buffer, const int nread, const int stride) {
//   int a;
//   for (int t=threadIdx.x + blockIdx.x*blockDim.x;t < nread;t+=blockDim.x*gridDim.x) {
//     a = buffer[t*stride];
//   }
// }

template <typename T>
__global__ void memoryWriteKernel(T* buffer, const int nwrite, const int stride, const int offset) {
  // for (int t=threadIdx.x + blockIdx.x*blockDim.x;t < nwrite;t+=blockDim.x*gridDim.x) {
    int t = threadIdx.x + blockIdx.x*blockDim.x;
    int wid = t / 32;
    int tid = t % 32;
    // T a = (t*nwrite*blockDim.x) + nwrite*threadIdx.x;
    // buffer[t*stride + offset] = a;
    buffer[wid*stride + tid + offset] = 1;
  // }
}

__global__ void memoryWriteKernel2(char* buffer, const int nwrite, const int stride, const int offset) {
  int t = threadIdx.x + blockIdx.x*blockDim.x;
  int wid = t / 32;
  int tid = t % 32;
  if (tid == 0) buffer[wid*stride + offset] = 1;
}

__global__ void cyclesPerOperationKernel() {
  // int a = threadIdx.x;
  // int b = blockIdx.x;
  long long int start = clock64();
  int a = threadIdx.x;
  int b = blockIdx.x;
  b += a;
  a *= 17;
  b += 3;
  a -= b;
  long long int end = clock64();
  printf("threadIdx.x %d cycles %lld a %d b %d\n", threadIdx.x, end-start, a, b);
}

__global__ void cacheLineKernel(double* buffer, double* res) {
  int t = threadIdx.x;
  //
  double a = buffer[t+1];
  //
  double sum = 0.0;
  for (int i=0;i < 32;i++) {
    sum += __shfl(a, i);
  }
  if (threadIdx.x == 0) res[0] = sum;
}

// ############################################################################
// ############################################################################
// ############################################################################

static int SM_major = 0;

template <typename T> void pChase(int stride);
void memoryTransactions();
template <typename T> void memoryLatency(int nwarp, int nsm);
void clearCache(int* buffer, const int bufferSize);
void cyclesPerOperation();
template <typename T> void memoryWrite(int stride, int offset);
void memoryWrite2(int stride, int offset);
void cacheLine();
void printDeviceInfo();

int main(int argc, char *argv[]) {

  int stride = 1;
  int offset = 0;
  int deviceID = 0;
  bool arg_ok = true;
  if (argc >= 2) {
    int i = 1;
    while (i < argc) {
      if (strcmp(argv[i], "-stride") == 0) {
        sscanf(argv[i+1], "%d", &stride);
        i += 2;
      } else if (strcmp(argv[i], "-offset") == 0) {
        sscanf(argv[i+1], "%d", &offset);
        i += 2;
      } else if (strcmp(argv[i], "-device") == 0) {
        sscanf(argv[i+1], "%d", &deviceID);
        i += 2;
      } else {
        arg_ok = false;
        break;
      }
    }
  } else if (argc > 1) {
    arg_ok = false;
  }

  if (!arg_ok) {
    printf("cumb [options]\n");
    printf("Options:\n");
    printf("-stride [stride]\n");
    printf("-offset [offset]\n");
    printf("-device [device]\n");
    return 1;
  }

  cudaCheck(hipSetDevice(deviceID));
  printDeviceInfo();

  int* buffer = NULL;
  int bufferSize = 1000000;
  allocate_device<int>(&buffer, bufferSize);

  // for (int i=1;i <= 1;i++) {
  //   clearCache(buffer, bufferSize);
  //   memoryLatency<long long int>(i, 1);
  // }

  // clearCache(buffer, bufferSize);
  // memoryLatency<int>(1);

  // clearCache(buffer, bufferSize);
  // memoryWrite2(stride, offset);

  // clearCache(buffer, bufferSize);
  // memoryTransactions();

  if (stride == 0) {
    for (int i=1;i <= 32;i++) {
      clearCache(buffer, bufferSize);
      pChase<int>(i);
    }
  } else {
    clearCache(buffer, bufferSize);
    pChase<int>(stride);
  }

  // clearCache(buffer, bufferSize);
  // pChase<long long int>(stride);

  // clearCache(buffer, bufferSize);
  // memoryWrite<long long int>(stride, offset);

  // clearCache(buffer, bufferSize);
  // cacheLine();

  deallocate_device<int>(&buffer);

  // cyclesPerOperation();

  cudaCheck(hipDeviceReset());
  return 0;
}

template <typename T>
void pChase(int stride) {
  int nthread = stride;
  int arraySize = 320*1024*1024/sizeof(T);
  T* array;
  allocate_device<T>(&array, arraySize);
  T* h_array = new T[arraySize];
  for (int i=0;i < arraySize;i++) {
    h_array[i] = (T)(-1);
  }
  // for (int i=0;i < arraySize;i++) {
  //   int iblock = i/nthread;
  //   int ithread = i % nthread;
  //   h_array[i] = (T)((iblock + stride)*nthread + ithread) % arraySize;
  // }
  for (int i=0;i < arraySize;i+=32) {
    h_array[i] = (32*nthread + i) % arraySize;
  }
  // int k = 0;
  // for (int j=0;j < 33;j++) {
  //   for (int i=0;i < 32;i++) {
  //     printf("%d ", (int)h_array[k++]);
  //   }
  //   printf("\n");
  // }
  copy_HtoD_sync<T>(h_array, array, arraySize);
  cudaCheck(hipDeviceSynchronize());
  delete [] h_array;

  if (SM_major >= 5) {
    pChaseMaxwellKernel<T> <<< 1, nthread, 320*sizeof(T) >>>(array, 320);
  } else {
    pChaseKernel<T, 320> <<< 1, nthread >>>(array);
  }
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());

  deallocate_device<T>(&array);
}

void cacheLine() {
  double* buffer = NULL;
  allocate_device<double>(&buffer, 256);

  cudaCheck(hipDeviceSynchronize());

  int nthread = 32;
  int nblock = 1;
  cacheLineKernel <<< nblock, nthread >>>(buffer, &buffer[128]);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());

  deallocate_device<double>(&buffer);
}

void memoryTransactions() {
  float* buffer = NULL;
  allocate_device<float>(&buffer, 1024);

  int nthread = 1;
  int nblock =1;
  memoryTransactionKernel<float> <<< nblock, nthread >>>(buffer);
  cudaCheck(hipGetLastError());

  memoryTransactionKernel<float> <<< nblock, nthread >>>(buffer);
  cudaCheck(hipGetLastError());

  memoryTransactionKernel<float> <<< nblock, nthread >>>(buffer);
  cudaCheck(hipGetLastError());

  memoryTransactionKernel<double> <<< nblock, nthread >>>((double *)buffer);
  cudaCheck(hipGetLastError());

  memoryTransactionKernel<double> <<< nblock, nthread >>>((double *)buffer);
  cudaCheck(hipGetLastError());

  memoryTransactionKernel<double> <<< nblock, nthread >>>((double *)buffer);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());  

  deallocate_device<float>(&buffer);
}

template <typename T>
void memoryWrite(int stride, int offset) {
  int nwrite = 31249408/2;
  int bufferSize = nwrite*34;
  T* buffer = NULL;
  allocate_device<T>(&buffer, bufferSize);
  printf("bufferSize %f GB\n", bufferSize*sizeof(T)/1000000000.0f);

  cudaCheck(hipDeviceSynchronize());

  int nthread = 512;
  int nblock = nwrite/nthread;
  int numActiveBlock;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock, memoryWriteKernel<T>, nthread, 0);
  printf("nthread %d nblock %d numActiveBlock %d\n", nthread, nblock, numActiveBlock);
  memoryWriteKernel<T> <<< nblock, nthread >>>(buffer, nwrite, stride, offset);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());

  long long int bytesWritten = nwrite*sizeof(T);
  printf("wrote %lld bytes using stride %d and offset %d\n", bytesWritten, stride, offset);

  deallocate_device<T>(&buffer);
}

void memoryWrite2(int stride, int offset) {
  int nwrite = 31249408/2;
  int bufferSize = nwrite*34;
  char* buffer = NULL;
  allocate_device<char>(&buffer, bufferSize);
  printf("bufferSize %f GB\n", bufferSize*sizeof(char)/1000000000.0f);

  cudaCheck(hipDeviceSynchronize());

  int nthread = 512;
  int nblock = nwrite/nthread;
  int numActiveBlock;
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&numActiveBlock, memoryWriteKernel2, nthread, 0);
  printf("nthread %d nblock %d numActiveBlock %d\n", nthread, nblock, numActiveBlock);
  memoryWriteKernel2 <<< nblock, nthread >>>(buffer, nwrite, stride, offset);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());

  printf("wrote using stride %d and offset %d\n", stride, offset);

  deallocate_device<char>(&buffer);
}

template <typename T> void memoryLatency(int nwarp, int nsm) {

  T* bufferIn = NULL;
  T* bufferOut = NULL;
  allocate_device<T>(&bufferIn, 16384*nwarp*nsm);
  allocate_device<T>(&bufferOut, 16384*nwarp*nsm);

  cudaCheck(hipDeviceSynchronize());

  // printf("%d\n", nwarp);
  // int nthread = 32*nwarp;
  // int nblock = nsm;
  // int shmemsize = nthread*sizeof(int);
  // memoryLatencyKernel<T> <<< nblock, nthread, shmemsize >>>(bufferIn, bufferOut);
  // cudaCheck(hipGetLastError());

  int nthread = 32*nwarp;
  int nblock = nsm;
  int shmemsize = nthread*sizeof(int);
  memoryLatencyKernel2<T> <<< nblock, nthread, shmemsize >>>(bufferOut);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());
  // printf("\n");

  deallocate_device<T>(&bufferIn);
  deallocate_device<T>(&bufferOut);

}

void clearCache(int* buffer, const int bufferSize) {
  cudaCheck(hipDeviceSynchronize());

  int nthread = 1024;
  int nblock = (bufferSize - 1)/nthread + 1;
  clearCacheKernel <<< nblock, nthread >>>(buffer, bufferSize);
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());
}

void cyclesPerOperation() {
  cudaCheck(hipDeviceSynchronize());

  int nthread = 32;
  int nblock = 1;
  cyclesPerOperationKernel <<< nblock, nthread >>>();
  cudaCheck(hipGetLastError());

  cudaCheck(hipDeviceSynchronize());  
}

void printDeviceInfo() {
  int deviceID;
  cudaCheck(hipGetDevice(&deviceID));
  hipDeviceProp_t prop;
  cudaCheck(hipGetDeviceProperties(&prop, deviceID));
  hipSharedMemConfig pConfig;
  cudaCheck(hipDeviceGetSharedMemConfig(&pConfig));
  int shMemBankSize = 4;
  if (pConfig == hipSharedMemBankSizeEightByte) shMemBankSize = 8;
  double mem_BW = (double)(prop.memoryClockRate*2*(prop.memoryBusWidth/8))/1.0e6;
  SM_major = prop.major;
  printf("Using %s SM version %d.%d\n", prop.name, prop.major, prop.minor);
  printf("Clock %1.3lfGhz numSM %d ECC %d mem BW %1.2lfGB/s shMemBankSize %dB\n", (double)prop.clockRate/1e6,
	 prop.multiProcessorCount, prop.ECCEnabled, mem_BW, shMemBankSize);
  printf("L2 %1.2lfMB\n", (double)prop.l2CacheSize/(double)(1024*1024));
}
