/******************************************************************************
MIT License

Copyright (c) 2016 Antti-Pekka Hynninen
Copyright (c) 2016 Oak Ridge National Laboratory (UT-Batelle)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*******************************************************************************/

#include <stdio.h>
#include "CudaUtils.h"

//----------------------------------------------------------------------------------------

void clear_device_array_async_T(void *data, const int ndata, hipStream_t stream, const size_t sizeofT) {
  cudaCheck(hipMemsetAsync(data, 0, sizeofT*ndata, stream));
}

void clear_device_array_T(void *data, const int ndata, const size_t sizeofT) {
  cudaCheck(hipMemset(data, 0, sizeofT*ndata));
}

//----------------------------------------------------------------------------------------
//
// Allocate gpu memory
// pp = memory pointer
// len = length of the array
//
void allocate_device_T(void **pp, const int len, const size_t sizeofT) {
  cudaCheck(hipMalloc(pp, sizeofT*len));
}

//----------------------------------------------------------------------------------------
//
// Deallocate gpu memory
// pp = memory pointer
//
void deallocate_device_T(void **pp) {
  
  if (*pp != NULL) {
    cudaCheck(hipFree((void *)(*pp)));
    *pp = NULL;
  }

}

//----------------------------------------------------------------------------------------
//
// Copies memory Host -> Device
//
void copy_HtoD_async_T(const void *h_array, void *d_array, int array_len, hipStream_t stream,
           const size_t sizeofT) {
  cudaCheck(hipMemcpyAsync(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice, stream));
}

void copy_HtoD_T(const void *h_array, void *d_array, int array_len,
     const size_t sizeofT) {
  cudaCheck(hipMemcpy(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice));
}

//----------------------------------------------------------------------------------------
//
// Copies memory Device -> Host
//
void copy_DtoH_async_T(const void *d_array, void *h_array, const int array_len, hipStream_t stream,
           const size_t sizeofT) {
  cudaCheck(hipMemcpyAsync(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost, stream));
}

void copy_DtoH_T(const void *d_array, void *h_array, const int array_len, const size_t sizeofT) {
  cudaCheck(hipMemcpy(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost));
}

